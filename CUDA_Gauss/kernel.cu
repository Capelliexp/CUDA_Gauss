//#include "hip/hip_runtime.h"
//#include ""
//#include "CPU_Functions.cu"

#include "project.cuh"

int main(int argc, char *argv[]){
	float** matrix = new float*[COLUMN_LENGTH];
	for (int i = 0; i < 4; i++)
		matrix[i] = new float[ROW_LENGTH];
	float* vector = new float[COLUMN_LENGTH];

	FillMatrixStandard(matrix, vector);

	SortCPU(matrix, vector);

	getchar();
	return 0;
}

void FillMatrixStandard(float** matrix, float* vector) {
	matrix[0][0] = 1.f;
	matrix[0][1] = 3.f;
	matrix[0][2] = 1.f;
	matrix[0][3] = 3.f;
					
	matrix[1][0] = 4.f;
	matrix[1][1] = -2.f;
	matrix[1][2] = -3.f;
	matrix[1][3] = 1.f;
					
	matrix[2][0] = 2.f;
	matrix[2][1] = 1.f;
	matrix[2][2] = -1.f;
	matrix[2][3] = -1.f;
					
	matrix[3][0] = 1.f;
	matrix[3][1] = 2.f;
	matrix[3][2] = -1.f;
	matrix[3][3] = -2.f;

	vector[0] = 14.f;
	vector[1] = 20.f;
	vector[2] = 9.f;
	vector[3] = 3.f;
}

void FillMAtrixRandom(float** matrix, float* vector) {}

void SortCPU(float** matrix, float* vector) {
	ForwardSubstitution(matrix, vector);
	BackSubstitution(matrix, vector);
}