#include "hip/hip_runtime.h"
#include "project.cuh"

__global__
void DeviceGaussForward(float* d_m, float* d_v) {
	int id = threadIdx.x + blockIdx.x * blockDim.x + 1;

	if (id < COLUMN_LENGTH) {
		int i = 0, factor;

		__shared__ float pivotRHS;
		__shared__ float pivotLHS[ROW_LENGTH];
		
		float RHS = d_v[id];
		float LHS[ROW_LENGTH] = { 1.f, 2.f, 3.f, 4.f };
		for (int j = 0; j < ROW_LENGTH; ++j) {
			int pos = id*ROW_LENGTH + j;
			LHS[j] = d_m[pos];
		}

		while (id > i) {
			__syncthreads();
			pivotRHS = d_v[i];
			for (int j = 0; j < ROW_LENGTH; ++j) pivotLHS[j] = d_m[i*ROW_LENGTH + j];

			factor = (LHS[i] / pivotLHS[i]) * (-1);

			for (int j = 0; j < ROW_LENGTH; j++) {
				LHS[j] += (factor * pivotLHS[j]);
			}
			RHS += (factor * pivotRHS);

			++i;
		}
		for (int j = 0; j < ROW_LENGTH; ++j) d_m[ROW_LENGTH*id + j] = LHS[j];
		d_v[id] = RHS;
	}
}

//__device__
//void FloatArrayCpy(float* dst, float* src, int row) {
//	for (int i = 0; i < ROW_LENGTH; ++i)
//		dst[i] = src[row*COLUMN_LENGTH + i];
//}

void InitCUDA(float** m, float* v, float* a) {
	float* d_m = nullptr;
	float* d_v = nullptr;
	float* d_a = nullptr;


	float* cuda_m = new float[ROW_LENGTH*COLUMN_LENGTH];
	for (int i = 0; i < COLUMN_LENGTH; i++){
		for (int j = 0; j < ROW_LENGTH; j++) {
			cuda_m[i * ROW_LENGTH + j] = m[i][j];
		}
	}


	cuErrorCheck(hipSetDevice(0));

	//Allocate memory
	cuErrorCheck(hipMalloc((void**)&d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_v, COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_a, COLUMN_LENGTH * sizeof(float)));

	cuErrorCheck(hipMemcpy(d_m, cuda_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_v, v, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_a, a, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));

	DeviceGaussForward<<<1, 4>>>(d_m, d_v);
	cuErrorCheck(hipGetLastError());

	cuErrorCheck(hipMemcpy(cuda_m, d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(v, d_v, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < ROW_LENGTH * COLUMN_LENGTH; i++) {
		if (i%ROW_LENGTH == 0) std::cout << std::endl;
		std::cout << cuda_m[i] << " ";
	}

	hipFree(d_m);
	hipFree(d_v);
	hipFree(d_a);
}

void cuErrorCheck(hipError_t cs)
{
	if (cs != hipSuccess)
		fprintf(stderr, "CUDA ERROR: %s\n", hipGetErrorString(cs));
}