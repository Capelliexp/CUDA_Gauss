#include "hip/hip_runtime.h"
#include "project.cuh"

__global__
void DeviceGaussForward(float** d_m, float* d_v) {
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	++id;
	int i = 0;

	__shared__ float pivotLHS[ROW_LENGTH];
	__shared__ float pivotRHS;
	float LHS[ROW_LENGTH];
	float RHS = d_v[id];

	memcpy(LHS, d_m[id], sizeof(float)*ROW_LENGTH);

	while (id > i && id < COLUMN_LENGTH) {
		memcpy(pivotLHS, d_m[i], sizeof(float)*ROW_LENGTH);
		pivotRHS = d_v[i];
		float factor = (LHS[i] / pivotLHS[i]) * (-1);

		for (int j = 0; j < ROW_LENGTH; j++) {
			LHS[j] += (factor * pivotLHS[j]);
		}
		RHS += (factor * pivotRHS);

		++i;
		__syncthreads();
	}
}

void InitCUDA(float** m, float* v, float* a) {
	float** d_m = nullptr;
	float* d_v = nullptr;
	float* d_a = nullptr;

	cuErrorCheck(hipSetDevice(0));

	//Allocate memory
	cuErrorCheck(hipMalloc((void**)&d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_v, COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_a, COLUMN_LENGTH * sizeof(float)));

	cuErrorCheck(hipMemcpy(d_m, m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_v, v, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_a, a, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));



	hipFree(d_m);
	hipFree(d_v);
	hipFree(d_a);
}

void cuErrorCheck(hipError_t cs)
{
	if (cs != hipSuccess)
		fprintf(stderr, "CUDA ERROR: %s\n", hipGetErrorString(cs));
}