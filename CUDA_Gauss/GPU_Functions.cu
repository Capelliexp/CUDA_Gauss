#include "hip/hip_runtime.h"
#include "project.cuh"

__global__
void DeviceGaussForwardLower(float* d_m, float* d_v) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if ((id *= ELEMENTS_PER_THREAD)< COLUMN_LENGTH) {
		++id;
		int i = 0;
		double factor;

		__shared__ double pivotRHS;
		__shared__ double pivotLHS[ROW_LENGTH];

		double RHS[ELEMENTS_PER_THREAD];
		double LHS[ELEMENTS_PER_THREAD][ROW_LENGTH];

		for (int j = 0; j < ELEMENTS_PER_THREAD; ++j) {
			float value = d_v[id + j];
			RHS[j] = value;
		}

		for (int k = 0; k < ELEMENTS_PER_THREAD; ++k) {
			for (int j = 0; j < ROW_LENGTH; ++j) {
				int pos = ((id+k) * ROW_LENGTH) + j;	//fel?
				float value = d_m[pos];
				LHS[k][j] = value;
			}
		}

		while ((id+ELEMENTS_PER_THREAD) > i) {
			__syncthreads();

			pivotRHS = d_v[i];

			for (int j = 0; j < ROW_LENGTH; ++j) {
				float value = d_m[i*ROW_LENGTH + j];
				pivotLHS[j] = value;
			}

			for (int g = 0; g < ELEMENTS_PER_THREAD; ++g) {
				if ((id + g) > i) {
					float a = LHS[g][i];
					float b = pivotLHS[i];
					float c = (a / b);
					factor = c * (-1);
					//factor = (LHS[g][i] / pivotLHS[i]) * (-1);

					for (int j = i; j < ROW_LENGTH; ++j) {
						float value = LHS[g][j] + (factor * pivotLHS[j]);
						LHS[g][j] = value;
					}
					float value = RHS[g] + (factor * pivotRHS);
					RHS[g] = value;
				}

				for (int j = 0; j < ROW_LENGTH; ++j) {
					d_m[ROW_LENGTH*(id + g) + j] = (abs(LHS[g][j]) < 0.001 ? 0 : LHS[g][j]);
				}
				d_v[id + g] = (abs(RHS[g]) < 0.001 ? 0 : RHS[g]);

				/*if (!((id + g) > (i-1))) {	//FLYTTA IFSATSEN �VERS I WHILE OCH TESTA D�. K�R SYNC EFTER IF
					
				}*/
			}
			++i;
		}
	}
}

__global__
void DeviceGaussForwardUpper(float* d_m, float* d_v, float* d_a) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if ((id *= ELEMENTS_PER_THREAD) < COLUMN_LENGTH) {
		int i = COLUMN_LENGTH - 1;
		double factor;

		__shared__ double pivotRHS;
		__shared__ double pivotLHS[ROW_LENGTH];

		double RHS[ELEMENTS_PER_THREAD];
		double LHS[ELEMENTS_PER_THREAD][ROW_LENGTH];

		for (int j = 0; j < ELEMENTS_PER_THREAD; ++j) {
			RHS[j] = d_v[id+j];
		}
		
		for (int k = 0; k < ELEMENTS_PER_THREAD; ++k) {
			for (int j = 0; j < ROW_LENGTH; ++j) {
				int pos = (k*ROW_LENGTH)+(id * ROW_LENGTH) + j;
				double a = d_m[pos];
				LHS[k][j] = a;
				//printf("id %d: LHS[%d] = %f \n",id, j, LHS[j]);
			}
		}

		while (id < i) {
			__syncthreads();

			pivotRHS = d_v[i];

			for (int j = 0; j < ROW_LENGTH; ++j) {
				pivotLHS[j] = d_m[i*ROW_LENGTH + j];
			}

			for (int g = 0; g < ELEMENTS_PER_THREAD; ++g) {
				if (id+g < i) {
					double a = LHS[g][i];	//obs! antar att vi jobbar med square matrix
					double b = pivotLHS[i];
					double c = (a / b);
					factor = c * (-1);
					//factor = (LHS[i] / pivotLHS[i]) * (-1);

					//if (id == 0) printf("factor: %f \n", factor);
					for (int j = 0; j < ROW_LENGTH; j++) {
						LHS[g][j] += (factor * pivotLHS[j]);
						//if (id == 0) printf("i %d: LHS[%d] = %f \n",i, j, LHS[j]);
					}
					//printf("\n");

					float piv = pivotRHS;
					float pivAndFac = piv*factor;
					float RightHandSideOfG = RHS[g];
					float value = RightHandSideOfG + pivAndFac;
					RHS[g] = value;
				}

				for (int j = 0; j < ROW_LENGTH; ++j) {
					int pos = ROW_LENGTH*(id + g) + j;
					float value = LHS[g][j];
					d_m[pos] = value;	//fel?
				}
				float value1 = RHS[g];
				d_v[id + g] = value1;

				//float denominator = LHS[g][i];
				//float value2 = RHS[g] / denominator;	//antagligen r�tt...
				//d_a[id + g] = value2;

				//float denominator = d_m[(g*ROW_LENGTH) + (id*ROW_LENGTH) + i];
				//float value2 = d_v[id + g] / denominator;	//antagligen r�tt...
				//d_a[id + g] = value2;

				d_a[id + g] = d_v[id + g] / d_m[(id + g)*ROW_LENGTH + (g == 0 ? i-1 : i)];
				/*if (!(id+g < i)) {
					
				}*/
			}
			--i;
		}

		if (id == COLUMN_LENGTH - 1) {
			d_a[id] = RHS[0] / LHS[0][ROW_LENGTH - 1];	//fel? Nej, r�tt?
		}

	}
}

void InitCUDA(float** m, float* v, float* a) {
	float* d_m = nullptr;
	float* d_v = nullptr;
	float* d_a = nullptr;

	float duration = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* cuda_m = new float[ROW_LENGTH*COLUMN_LENGTH];
	for (int i = 0; i < COLUMN_LENGTH; i++){
		for (int j = 0; j < ROW_LENGTH; j++) {
			cuda_m[i * ROW_LENGTH + j] = m[i][j];
		}
	}

	cuErrorCheck(hipSetDevice(0));

	//Allocate memory
	cuErrorCheck(hipMalloc((void**)&d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_v, COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_a, COLUMN_LENGTH * sizeof(float)));

	//Copy memory
	cuErrorCheck(hipMemcpy(d_m, cuda_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_v, v, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_a, a, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));

	//hipEventRecord(start);
	DeviceGaussForwardLower<<<NR_OF_BLOCKS, THREADS_PER_BLOCK>>>(d_m, d_v);	//upper
	cuErrorCheck(hipGetLastError());
	//hipEventRecord(stop);

	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&duration, start, stop);
	//std::cout << "CUDA 1 time: " << duration / 1000 << " sec" << std::endl;

	//Copy memory
	cuErrorCheck(hipMemcpy(cuda_m, d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(v, d_v, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));

	//PrintMatrix("After lower", cuda_m, v, a);

	//hipEventRecord(start);
	DeviceGaussForwardUpper<<<NR_OF_BLOCKS, THREADS_PER_BLOCK>>>(d_m, d_v, d_a);	//lower
	cuErrorCheck(hipGetLastError());
	//hipEventRecord(stop);

	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&duration, start, stop);
	//std::cout << "CUDA 2 time: " << duration / 1000 << " sec" << std::endl;

	//Copy memory
	cuErrorCheck(hipMemcpy(cuda_m, d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(v, d_v, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(a, d_a, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));

	//PrintMatrix("After upper", cuda_m, v, a);

	std::cout << "GPU solution:" << std::endl;
	for (int i = 0; i < COLUMN_LENGTH-1; i++) {
		std::cout << a[i] << ", ";
	}
	std::cout << a[COLUMN_LENGTH - 1] << std::endl << std::endl;

	//Free memory
	hipFree(d_m);
	hipFree(d_v);
	hipFree(d_a);
}

void cuErrorCheck(hipError_t cs)
{
	if (cs != hipSuccess)
		fprintf(stderr, "CUDA ERROR: %s\n", hipGetErrorString(cs));
}