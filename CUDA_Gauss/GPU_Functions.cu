#include "hip/hip_runtime.h"
#include "project.cuh"

__global__
void DeviceGaussForwardLower(float* d_m, float* d_v) {
	int id = threadIdx.x + blockIdx.x * blockDim.x + 1;

	if (id < COLUMN_LENGTH) {
		int i = 0;
		double factor;

		__shared__ double pivotRHS;
		__shared__ double pivotLHS[ROW_LENGTH];

		double RHS = d_v[id];
		double LHS[ROW_LENGTH];

		for (int j = 0; j < ROW_LENGTH; ++j) {
			int pos = (id * ROW_LENGTH) + j;
			LHS[j] = d_m[pos];
		}

		while (id > i) {
			__syncthreads();
			pivotRHS = d_v[i];
			for (int j = 0; j < ROW_LENGTH; ++j) pivotLHS[j] = d_m[i*ROW_LENGTH + j];

			factor = (LHS[i] / pivotLHS[i]) * (-1);

			for (int j = i; j < ROW_LENGTH; j++) LHS[j] += (factor * pivotLHS[j]);
			RHS += (factor * pivotRHS);

			++i;

			if (id < COLUMN_LENGTH) {
				for (int j = 0; j < ROW_LENGTH; ++j) {
					d_m[ROW_LENGTH*id + j] = LHS[j];
				}
				d_v[id] = RHS;
			}
		}
	}
}

__global__
void DeviceGaussForwardUpper(float* d_m, float* d_v, float* d_a) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id < COLUMN_LENGTH) {
		int i = COLUMN_LENGTH - 1;
		double factor;

		__shared__ double pivotRHS;
		__shared__ double pivotLHS[ROW_LENGTH];

		double RHS = d_v[id];
		double LHS[ROW_LENGTH];

		for (int j = 0; j < ROW_LENGTH; ++j) {
			int pos = (id * ROW_LENGTH) + j;
			double a = d_m[pos];
			LHS[j] = a;
			//printf("id %d: LHS[%d] = %f \n",id, j, LHS[j]);
		}

		int iterator = 0;

		if (id == COLUMN_LENGTH-1) {
			d_a[id] = RHS / LHS[ROW_LENGTH-1];
		}

		while (id < i) {
			__syncthreads();
			pivotRHS = d_v[i];

			for (int j = 0; j < ROW_LENGTH; ++j) {
				pivotLHS[j] = d_m[i*ROW_LENGTH + j];
			}

			double a = LHS[i];
			double b = pivotLHS[i];
			double c = (a / b);
			factor = c * (-1);
			//factor = (LHS[i] / pivotLHS[i]) * (-1);

			//if (id == 0) printf("factor: %f \n", factor);
			for (int j = 0; j < ROW_LENGTH; j++) {
				LHS[j] += (factor * pivotLHS[j]);
				//if (id == 0) printf("i %d: LHS[%d] = %f \n",i, j, LHS[j]);
			}
			//printf("\n");

			RHS += (factor * pivotRHS);

			--i;
			++iterator;

			if (!(id < i)) {
				for (int j = 0; j < ROW_LENGTH; ++j) {
					d_m[ROW_LENGTH*id + j] = LHS[j];
				}
				d_v[id] = RHS;
				d_a[id] = RHS / LHS[i];
			}
			
		}
	}
}

void InitCUDA(float** m, float* v, float* a) {
	float* d_m = nullptr;
	float* d_v = nullptr;
	float* d_a = nullptr;

	float duration = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* cuda_m = new float[ROW_LENGTH*COLUMN_LENGTH];
	for (int i = 0; i < COLUMN_LENGTH; i++){
		for (int j = 0; j < ROW_LENGTH; j++) {
			cuda_m[i * ROW_LENGTH + j] = m[i][j];
		}
	}

	cuErrorCheck(hipSetDevice(0));

	//Allocate memory
	cuErrorCheck(hipMalloc((void**)&d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_v, COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_a, COLUMN_LENGTH * sizeof(float)));

	//Copy memory
	cuErrorCheck(hipMemcpy(d_m, cuda_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_v, v, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_a, a, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));

	//hipEventRecord(start);
	DeviceGaussForwardLower<<<1, 5>>>(d_m, d_v);	//upper
	cuErrorCheck(hipGetLastError());
	//hipEventRecord(stop);

	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&duration, start, stop);
	//std::cout << "CUDA 1 time: " << duration / 1000 << " sec" << std::endl;

	//Copy memory
	cuErrorCheck(hipMemcpy(cuda_m, d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(v, d_v, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));

	PrintMatrix(cuda_m, v, a);

	//hipEventRecord(start);
	DeviceGaussForwardUpper<<<1, 5>>>(d_m, d_v, d_a);	//lower
	cuErrorCheck(hipGetLastError());
	//hipEventRecord(stop);

	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&duration, start, stop);
	//std::cout << "CUDA 2 time: " << duration / 1000 << " sec" << std::endl;

	//Copy memory
	cuErrorCheck(hipMemcpy(cuda_m, d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(v, d_v, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(a, d_a, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));

	PrintMatrix(cuda_m, v, a);

	//Free memory
	hipFree(d_m);
	hipFree(d_v);
	hipFree(d_a);
}

void cuErrorCheck(hipError_t cs)
{
	if (cs != hipSuccess)
		fprintf(stderr, "CUDA ERROR: %s\n", hipGetErrorString(cs));
}