#include "hip/hip_runtime.h"
#include "project.cuh"

__global__
void DeviceGaussForwardLower(float* d_m, float* d_v) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if ((id *= ELEMENTS_PER_THREAD)< COLUMN_LENGTH) {
		++id;
		int i = 0;
		double factor;

		__shared__ double pivotRHS;
		__shared__ double pivotLHS[ROW_LENGTH];

		double RHS[ELEMENTS_PER_THREAD];
		double LHS[ELEMENTS_PER_THREAD][ROW_LENGTH];

		for (int j = 0; j < ELEMENTS_PER_THREAD; ++j) {
			RHS[j] = d_v[id + j];
		}

		for (int k = 0; k < ELEMENTS_PER_THREAD; ++k) {
			for (int j = 0; j < ROW_LENGTH; ++j) {
				LHS[k][j] = d_m[((id + k) * ROW_LENGTH) + j];
			}
		}

		while ((id+ELEMENTS_PER_THREAD) > i) {
			__syncthreads();

			pivotRHS = d_v[i];

			for (int j = 0; j < ROW_LENGTH; ++j) {
				pivotLHS[j] = d_m[i*ROW_LENGTH + j];
			}

			for (int g = 0; g < ELEMENTS_PER_THREAD; ++g) {
				if ((id + g) > i) {
					factor = (LHS[g][i] / pivotLHS[i]) * (-1);

					for (int j = i; j < ROW_LENGTH; ++j) {
						LHS[g][j] = LHS[g][j] + (factor * pivotLHS[j]);
					}
					RHS[g] += (factor * pivotRHS);
				}

				if (id < (i+(ELEMENTS_PER_THREAD)+g )) {	//MAGIC
					for (int j = 0; j < ROW_LENGTH; ++j) {
						d_m[ROW_LENGTH*(id + g) + j] = (abs(LHS[g][j]) < 0.001 ? 0 : LHS[g][j]);
					}
					d_v[id + g] = (abs(RHS[g]) < 0.001 ? 0 : RHS[g]);
				}
			}
			++i;
		}
	}
}

__global__
void DeviceGaussForwardUpper(float* d_m, float* d_v, float* d_a) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if ((id *= ELEMENTS_PER_THREAD) < COLUMN_LENGTH) {
		int i = COLUMN_LENGTH - 1;
		double factor;

		__shared__ double pivotRHS;
		__shared__ double pivotLHS[ROW_LENGTH];

		double RHS[ELEMENTS_PER_THREAD];
		double LHS[ELEMENTS_PER_THREAD][ROW_LENGTH];

		for (int j = 0; j < ELEMENTS_PER_THREAD; ++j) {
			RHS[j] = d_v[id+j];
		}
		
		for (int k = 0; k < ELEMENTS_PER_THREAD; ++k) {
			for (int j = 0; j < ROW_LENGTH; ++j) {
				LHS[k][j] = d_m[(k*ROW_LENGTH) + (id * ROW_LENGTH) + j];
			}
		}

		while (id < i) {
			__syncthreads();

			pivotRHS = d_v[i];

			for (int j = 0; j < ROW_LENGTH; ++j) {
				pivotLHS[j] = d_m[i*ROW_LENGTH + j];
			}

			for (int g = 0; g < ELEMENTS_PER_THREAD; ++g) {
				if (id+g < i) {
					factor = (LHS[g][i] / pivotLHS[i]) * (-1);
					for (int j = 0; j < ROW_LENGTH; j++) {
						LHS[g][j] += (factor * pivotLHS[j]);
					}
					RHS[g] += (pivotRHS*factor);
				}

				if (id+g+1 > i-(ELEMENTS_PER_THREAD)) {	//MAGIC
					for (int j = 0; j < ROW_LENGTH; ++j) {
						d_m[ROW_LENGTH*(id + g) + j] = LHS[g][j];
					}
					d_v[id + g] = RHS[g];
					d_a[id + g] = d_v[id + g] / d_m[(id + g)*ROW_LENGTH + (g == 0 ? i - 1 : i)];
				}
			}
			--i;
		}
		if (id == COLUMN_LENGTH - 1) {
			d_a[id] = RHS[0] / LHS[0][ROW_LENGTH - 1];
		}

	}
}

void InitCUDA(float** m, float* v, float* a) {
	float* d_m = nullptr;
	float* d_v = nullptr;
	float* d_a = nullptr;

	float duration = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* cuda_m = new float[ROW_LENGTH*COLUMN_LENGTH];
	for (int i = 0; i < COLUMN_LENGTH; i++){
		for (int j = 0; j < ROW_LENGTH; j++) {
			cuda_m[i * ROW_LENGTH + j] = m[i][j];
		}
	}

	cuErrorCheck(hipSetDevice(0));

	//Allocate memory
	cuErrorCheck(hipMalloc((void**)&d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_v, COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_a, COLUMN_LENGTH * sizeof(float)));

	//Copy memory
	cuErrorCheck(hipMemcpy(d_m, cuda_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_v, v, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_a, a, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));

	hipEventRecord(start);	//clock start
	DeviceGaussForwardLower<<<1, THREADS_PER_BLOCK>>>(d_m, d_v);
	DeviceGaussForwardUpper<<<1, THREADS_PER_BLOCK>>>(d_m, d_v, d_a);
	hipEventRecord(stop);	//clock stop

	//time spent on CUDA gauss calc
	hipEventSynchronize(stop);
	hipEventElapsedTime(&duration, start, stop);
	std::cout << "CUDA time: " << duration / 1000 << " sec" << std::endl;

	//Copy memory
	cuErrorCheck(hipMemcpy(cuda_m, d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(v, d_v, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
	cuErrorCheck(hipMemcpy(a, d_a, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));

	/*std::cout << "GPU solution:" << std::endl;
	for (int i = 0; i < COLUMN_LENGTH-1; i++) {
		std::cout << a[i] << ", ";
	}
	std::cout << a[COLUMN_LENGTH - 1] << std::endl << std::endl;*/

	//Free memory
	hipFree(d_m);
	hipFree(d_v);
	hipFree(d_a);
}

void cuErrorCheck(hipError_t cs)
{
	if (cs != hipSuccess)
		fprintf(stderr, "CUDA ERROR: %s\n", hipGetErrorString(cs));
}