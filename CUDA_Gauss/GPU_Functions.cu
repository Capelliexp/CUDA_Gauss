#include "hip/hip_runtime.h"
#include "project.cuh"

__global__
void DeviceGaussForward(float* d_m/*, float* d_v*/) {
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	//++id;	//1-4
	int i = 0;	

	if (id < COLUMN_LENGTH) {
		printf("%f ", d_m[id]);
		__syncthreads();
		printf("\n");
		/*__shared__ float pivotLHS[ROW_LENGTH];
		__shared__ float pivotRHS;
		float LHS[ROW_LENGTH];
		float RHS = d_v[id];

		FloatArrayCpy(LHS, d_m[id], ROW_LENGTH);

		while (id > i) {
			FloatArrayCpy(pivotLHS, d_m[i], ROW_LENGTH);
			pivotRHS = d_v[i];
			float factor = (LHS[i] / pivotLHS[i]) * (-1);

			for (int j = 0; j < ROW_LENGTH; j++) {
				LHS[j] += (factor * pivotLHS[j]);
			}
			RHS += (factor * pivotRHS);

			++i;
			__syncthreads();
		}

		FloatArrayCpy(d_m[id], LHS, ROW_LENGTH);
		d_v[id] = RHS;*/
	}
}

__device__
void FloatArrayCpy(float* dst, float* src, int length) {
	for (int i = 0; i < length; ++i)
		dst[i] = src[i];
}

void InitCUDA(float** m, float* v, float* a) {
	float* d_m = nullptr;
	/*float* d_v = nullptr;
	float* d_a = nullptr;*/

	cuErrorCheck(hipSetDevice(0));

	//Allocate memory
	//cuErrorCheck(hipMalloc((void**)&d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float)));
	/*cuErrorCheck(hipMalloc((void**)&d_v, COLUMN_LENGTH * sizeof(float)));
	cuErrorCheck(hipMalloc((void**)&d_a, COLUMN_LENGTH * sizeof(float)));*/

	size_t pitch;
	hipMallocPitch(&d_m, &pitch, sizeof(float)*ROW_LENGTH, COLUMN_LENGTH);

	cuErrorCheck(hipMemcpy2D(d_m, pitch, m, sizeof(float)*COLUMN_LENGTH, sizeof(float)*ROW_LENGTH, COLUMN_LENGTH, hipMemcpyHostToDevice));
	/*cuErrorCheck(hipMemcpy(d_v, v, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));
	cuErrorCheck(hipMemcpy(d_a, a, COLUMN_LENGTH * sizeof(float), hipMemcpyHostToDevice));*/

	DeviceGaussForward<<<1, 4>>>(d_m/*, d_v*/);
	//cuErrorCheck(hipGetLastError());

	//cuErrorCheck(hipMemcpy(m, d_m, ROW_LENGTH * COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));

	//cuErrorCheck(hipMemcpy(v, d_v, COLUMN_LENGTH * sizeof(float), hipMemcpyDeviceToHost));



	hipFree(d_m);
	/*hipFree(d_v);
	hipFree(d_a);*/
}

void cuErrorCheck(hipError_t cs)
{
	if (cs != hipSuccess)
		fprintf(stderr, "CUDA ERROR: %s\n", hipGetErrorString(cs));
}